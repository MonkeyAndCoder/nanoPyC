
#include <hip/hip_runtime.h>
/* Logic
# original code:
for i in range(M):
    for j in range(N):
    	b[i,j] = exp(a[i,j])

# parallelized code:
nthreads = 128
for i in range(M):  # parallelized among thread blocks
    for j in range(N):  # parallelized among threads
    	b[i,j] = exp(a[i,j])

*/



extern "C" __global__
void kernel(int M, int N, float* a, float* b) {
    int m = blockDim.x * blockIdx.x + threadIdx.x;
    if (m > M) {
        return;
    }
    
    // Each thread works on an entire row
    for (int i = 0; i < N; i++) {
        b[m*N + i] = exp(a[m*N + i]);
    }
}