
#include <hip/hip_runtime.h>
extern "C" __global__
void kernel(int M, int N, float* a, float* b) {
    int m = blockDim.x * blockIdx.x + threadIdx.x;
    if (m > M) {
        return;
    }
    
    for (int i = 0; i < N; i++) {
        b[m*N + i] = exp(a[m*N + i]);
    }
}