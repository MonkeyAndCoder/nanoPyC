
#include <hip/hip_runtime.h>
extern "C" __global__
void kernel(int M, int N, float* a, float* b) {
    int m = blockDim.y * blockIdx.y + threadIdx.y;
    int n = blockDim.x * blockIdx.x + threadIdx.x;

    if (m > M || n > N) {
        return;
    }

    b[m*N+n] = exp(a[m*N+n]);
}