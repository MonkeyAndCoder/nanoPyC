
#include <hip/hip_runtime.h>
extern "C" __global__
void kernel(int M, int N, float* a, float* b) {
    int m = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < N; i++) {
        b[m*N + i] = a[m*N + i] + 1;
    }
}