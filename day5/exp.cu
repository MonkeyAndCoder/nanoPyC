
#include <hip/hip_runtime.h>
extern "C" __global__
void kernel(int M, int N, float* a, float* b, float* c) {
    int m = blockDim.y * blockIdx.y + threadIdx.y;
    int n = blockDim.x * blockIdx.x + threadIdx.x;

    if (m > M || n > N) {
        return;
    }

    // Neighboring threads will access contiguous memory locations now.
    c[m*N+n] = exp(a[m*N+n]) / b[m];
}